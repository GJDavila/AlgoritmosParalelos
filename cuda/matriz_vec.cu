
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
 using namespace std;


__global__ void MatrizVectorKernel(float *A, float* B, float* C, int n)
{
		int i=n*blockIdx.x;
		for(int j=0;j<n;j++)
		{
			C[blockIdx.x] += (A[i+j]*B[j]);
		}
}

void MatrizVector(float *A, float* B, float* C, int n)
{
        int size1 = n*n * sizeof(float);
				int size2 = n * sizeof(float);
        float *d_A, *d_B, *d_C;
        hipMalloc((void **) &d_A, size1);
        hipMemcpy(d_A, A, size1, hipMemcpyHostToDevice);
        hipMalloc((void **) &d_B, size2);
        hipMemcpy(d_B, B, size2, hipMemcpyHostToDevice);
        hipMalloc((void **) &d_C, size2);
        MatrizVectorKernel<<<n,1>>>(d_A, d_B, d_C, n);
        hipMemcpy(C, d_C, size2, hipMemcpyDeviceToHost);
        // Free device memory for A, B, C
        hipFree(d_A); hipFree(d_B); hipFree (d_C);
}


int main()
{
	float *A,*B,*C;
	int n=10;
	A=new float[n*n];
	B=new float[n];
	C=new float[n];
	for(int i=0;i<n*n;i++){A[i]=i;if(i<n){B[i]=i;}}

	MatrizVector(A,B,C,n);
	for(int i=0;i<n;i++)
	{
		cout<<C[i]<<" ";
	}

}
