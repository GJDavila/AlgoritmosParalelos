
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
 using namespace std;

 ///compilar
 ////usr/local/cuda/bin/nvcc matriz_vec.cu -o multi



///-----------funcion kernel
__global__ void MatrizVectorKernel(float *A, float* B, float* C, int n)
{
		int i=n*blockIdx.x;
    int tmp=0;
		for(int j=0;j<n;j++)
		{
		    tmp+= (A[i+j]*B[j]);
		}
    	C[blockIdx.x] =tmp;
}

void MatrizVector(float *A, float* B, float* C, int n)
{
        int size1 = n*n * sizeof(float);
				int size2 = n * sizeof(float);
        float *d_A, *d_B, *d_C;
        hipMalloc((void **) &d_A, size1);
        hipMemcpy(d_A, A, size1, hipMemcpyHostToDevice);
        hipMalloc((void **) &d_B, size2);
        hipMemcpy(d_B, B, size2, hipMemcpyHostToDevice);
        hipMalloc((void **) &d_C, size2);
        MatrizVectorKernel<<<n,1>>>(d_A, d_B, d_C, n);
        hipMemcpy(C, d_C, size2, hipMemcpyDeviceToHost);
        // Free device memory for A, B, C
        hipFree(d_A); hipFree(d_B); hipFree (d_C);
}


int main()
{
	float *A,*B,*C;
	int n=10;
	A=new float[n*n];
	B=new float[n];
	C=new float[n];
	for(int i=0;i<n*n;i++){A[i]=i;if(i<n){B[i]=i;}}

	MatrizVector(A,B,C,n);
	for(int i=0;i<n;i++)
	{
		cout<<C[i]<<" ";
	}

}
