#include "hip/hip_runtime.h"
#include<stdio.h>
#include<iostream>
|using namespace std;

///compilar
////usr/local/cuda/bin/nvcc summatriz.cu -o sum


///funciones kernels

 __global__ void  MatrizAddKernel_B(float *A, float *B, float* C, int n)
 {
                 int i=threadIdx.x + blockDim.x * blockIdx.x;
                  if(i<(n*n)) C[i]=A[i]+B[i];
 }

 __global__ void MatrizAddKernel_C(float *A, float* B, float* C, int n)
 {
 		int i=n*blockIdx.x;
 		for(int j=0;j<n;j++)
 		{
 			C[i+j]=A[i+j]+B[i+j];
 		}
 }

 __global__ void  MatrizAddKernel_D(float *A, float* B, float* C, int n)
 {
 		int i=blockIdx.x;
 		for(int j=0;j<n;j++)
 		{
 			C[i+n*j]=A[i+n*j]+B[i+n*j];
 		}
 }
 void MatrizAdd(float *A, float* B, float* C, int n)
 {
         int size = n*n * sizeof(float);
         float *d_A, *d_B, *d_C;
         hipMalloc((void **) &d_A, size);
         hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
         hipMalloc((void **) &d_B, size);
         hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
         hipMalloc((void **) &d_C, size);
         MatrizAddKernel_B<<<5,20>>>(d_A, d_B, d_C, n);
         //MatrizAddKernel_C<<<n,1>>>(d_A, d_B, d_C, n);
         //MatrizAddKernel_D<<<n,1>>>(d_A, d_B, d_C, n);
         hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
         // Free device memory for A, B, C
         hipFree(d_A); hipFree(d_B); hipFree (d_C);
 }
 int main()
 {
   float *A,*B,*C;
   int n=10;
   A=new float[n*n];
   B=new float[n*n];
   C=new float[n*n];
   for(int i=0;i<n*n;i++){A[i]=i;B[i]=i;}

   MatrizAdd(A,B,C,n);
   for(int i=0;i<n*n;i++)
   {
     cout<<C[i]<<" ";
   }

 }
